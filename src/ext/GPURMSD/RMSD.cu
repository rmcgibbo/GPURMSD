#include "hip/hip_runtime.h"
#include "kernel_rmsd.cu"
#include <assert.h>
#include <iostream>
#include "RMSD.hh"

using std::cout;

void Rmsd::print_params() {
    
    cout << "numAtoms: " << numAtoms_ << "\n";
    cout << "numConfs: " << numConfs_ << "\n";
    cout << "numBlocks: " << numBlocks_ << "\n";
    cout << "mem size: " << size_ << "\n";
    cout << "GPU mem capacity: " << capacity_ << "\n";
    cout << "GPU compute capability : " << compute_capability_ << "\n";

}

void Rmsd::set_rmsd_array(int numConfs, float *h_rmsds) {

    assert(numConfs == numConfs_);
    h_rmsds_ = h_rmsds;

    hipError_t t;
    t=hipMalloc((void **) &d_rmsds_, numConfs_*sizeof(float));
    assert(t == 0);
}

/*
void Rmsd::set_rot_mat_array(int numConfs, int RotMatSize, float *h_rot_mat) {

    assert(numConfs == numConfs_);
    assert(RotMatSize == 9);
    h_rot_mat_ = h_rot_mat;

    hipError_t t;
    t=hipMalloc((void **) &d_rot_mat_, numConfs_*sizeof(float)*9);
    assert(t == 0);

}
*/

/*
void Rmsd::set_only_device_rot_mat_array() {

    hipError_t t;
    t=hipMalloc((void **) &d_rot_mat_, numConfs_*sizeof(float)*9);
    assert(t == 0);

}
*/

void Rmsd::set_subset_flag_array(int numConfs, int *h_subset_flag) {

    assert(numConfs == numConfs_);
    h_subset_flag_ = h_subset_flag;

    hipError_t t;
    t=hipMalloc((void **) &d_subset_flag_, numConfs_*sizeof(float));
    assert(t == 0);

    hipMemcpy(d_subset_flag_, h_subset_flag_, numConfs_*sizeof(float), hipMemcpyHostToDevice);

}

void Rmsd::all_against_one_rmsd(int test_conf) {

    k_all_against_one_rmsd<<<numBlocks_,threadsperblock_>>>(numAtoms_, numConfs_, test_conf, d_X_, d_rmsds_, d_G_);

    hipMemcpy(h_rmsds_, d_rmsds_, numConfs_ * sizeof(float), hipMemcpyDeviceToHost);

}

void Rmsd::all_against_one_lprmsd(int test_conf) {

    assert(d_subset_flag_ != NULL);
    assert(h_subset_flag_ != NULL);

    k_all_against_one_lprmsd<<<numBlocks_,threadsperblock_>>>(numAtoms_, numConfs_, test_conf, d_X_, d_rmsds_, d_G_, d_subset_flag_);

    hipMemcpy(h_rmsds_, d_rmsds_, numConfs_ * sizeof(float), hipMemcpyDeviceToHost);

}

/*
void Rmsd::apply_rotation() {

    assert(d_rot_mat_ != NULL);

    k_rotate_all<<<numBlocks_, threadsperblock_>>>( numConfs_, numAtoms_, d_X_, d_rot_mat_);

    //hipMemcpy(h_X_, d_X_, size_, hipMemcpyDeviceToHost);

}
*/

Rmsd::Rmsd(int numAtoms, int numConfs, int numDimens, float* h_X) :
    numAtoms_(numAtoms), 
    numConfs_(numConfs), 
    numBlocks_(ceil( (float) numConfs_ / (float) threadsperblock_ )), 
    size_(numAtoms*numConfs*numDims*sizeof(float)),
    h_X_(h_X),
    d_subset_flag_(NULL),
    h_subset_flag_(NULL),
    compute_capability_(0),
    capacity_(0)
{
    
    assert(numDimens == 3);

    set_gpu_parameters();

    assert(h_X_ != NULL); 
    //allocate on the GPU
    hipError_t t;
    t=hipMalloc((void **) &d_X_, size_);
    assert(t == 0);

    t=hipMalloc((void **) &d_G_, numConfs_*sizeof(float));
    assert(t == 0);
    

    t=hipMemcpy(d_X_,h_X_,size_,hipMemcpyHostToDevice);
    assert(t == 0);
    assert(d_X_ != NULL);

    //check memory requirements and compute capability
    assert(size_ <= capacity_ );
    assert(compute_capability_>= 2.0);

}

void Rmsd::center_and_precompute_G() {

    assert(h_rmsds_ != NULL);
    assert(d_rmsds_ != NULL);

    center_conformers();
    precompute_G();

}

void Rmsd::set_gpu_parameters() {

    // this also does rotation
 
     hipDeviceProp_t prop;
     hipGetDeviceProperties( &prop, 0 );
     capacity_ = prop.totalGlobalMem;
     compute_capability_ = prop.major + 0.1*(float)prop.minor;

}

void Rmsd::precompute_G() {

    // kernel automatically takes care of conditions when
    // d_subset_flag_ is false
    k_precompute_G<<<numBlocks_, threadsperblock_>>>(numConfs_,numAtoms_,d_X_, d_G_,d_subset_flag_);

    have_precomputed_G_ = true;

}
 
void Rmsd::center_conformers() {

    k_center_conformers<<<numBlocks_, threadsperblock_>>>(numConfs_,numAtoms_,d_X_,d_subset_flag_);

    have_centered_ = true;

}

Rmsd::~Rmsd() {
    hipFree(d_X_);
    if( d_rmsds_ != NULL) {
    hipFree(d_rmsds_);
    }  
}
